
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

double cpu_time();

#define M 50
#define N 50

void calculate_solution_gold(double w[M][N], double epsilon, double diff)
{
    double ctime;
    double ctime1;
    double ctime2;
    int i;
    int j;
    int iterations;
    int iterations_print;

    double u[M][N];

    //  iterate until the  new solution W differs from the old solution U
    //  by no more than EPSILON.

    iterations = 0;
    iterations_print = 1;
    printf("\n");
    printf(" Iteration  Change\n");
    printf("\n");

    ctime1 = cpu_time();

    while (epsilon <= diff)
    {
        //  Save the old solution in U.

        for (i = 0; i < M; i++)
            for (j = 0; j < N; j++)
                u[i][j] = w[i][j];

        //  Determine the new estimate of the solution at the interior points.
        //  The new solution W is the average of north, south, east and west neighbors.

        diff = 0.0;
        for (i = 1; i < M - 1; i++)
        {
            for (j = 1; j < N - 1; j++)
            {
                w[i][j] = (u[i - 1][j] + u[i + 1][j] + u[i][j - 1] + u[i][j + 1]) / 4.0;

                if (diff < fabs(w[i][j] - u[i][j]))
                    diff = fabs(w[i][j] - u[i][j]);
            }
        }
        iterations++;
        if (iterations == iterations_print)
        {
            printf("  %8d  %lg\n", iterations, diff);
            iterations_print = 2 * iterations_print;
        }
    } //fin while epsilon

    ctime2 = cpu_time();
    ctime = ctime2 - ctime1;

    printf("\n");
    printf("  %8d  %lg\n", iterations, diff);
    printf("\n");
    printf("  Error tolerance achieved.\n");
    printf("  CPU time = %f\n", ctime);
}

double cpu_time()

//****************************************************************************80
//
//  Purpose:
//
//    CPU_TIME returns the current reading on the CPU clock.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license.
//
//  Modified:
//
//    06 June 2005
//
//  Author:
//
//    John Burkardt
//
//  Parameters:
//
//    Output, double CPU_TIME, the current reading of the CPU clock, in seconds.
//
{
    double value;

    value = (double)clock() / (double)CLOCKS_PER_SEC;

    return value;
}